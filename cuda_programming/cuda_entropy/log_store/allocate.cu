#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<time.h>
#include<math.h>
#include<fcntl.h>
#include<unistd.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>
#define BLOCK_SIZE 32


__global__ void entropy_kernel(int* input,float* output,float *logVal,int width,int height,int window_size){
    __shared__ int shared_mem[122];

    if(threadIdx.x==0){
        for(int i=0;i<122;i++){
            shared_mem[i]=logVal[i];
        }
    }

    __syncthreads();

    int x=blockIdx.x*blockDim.x+threadIdx.x;
    int y=blockIdx.y*blockDim.y+threadIdx.y;

    if(x<width && y<height){
        float entropy=0.0;

        int p[15]={0};
        for(int i=-window_size/2;i<window_size/2;i++){
            for(int j=-window_size/2;j<window_size/2;j++){
                int nx=x+i;
                int ny=y+j;

                if(nx>=0 && nx<width && ny>=0 && ny<height){
                    // use the value stored
                    p[input[ny*width+nx]]+=1;
                } 
            }
        }

        for(int i=0;i<15;i++){
            if(p[i]==0) continue;
            entropy+=(float)p[i]/(window_size*window_size)*(shared_mem[p[i]]-shared_mem[window_size*window_size]);
        }

        output[y*width+x]=-entropy;
    }
}

void allocate(int *array,float *entropy,int width,int height,int window_size){
    // value pass to GPU memory
    int *distributedArray=NULL;
    float *distributedLogVal=NULL,*logVal=NULL;
    float* distributedEntropyArray=NULL;

    // calculate size
    int arraySize=width*height*sizeof(int);
    int entropySize=width*height*sizeof(float);
    // max window size 121
    int logSize=122;

    // store the log value first
    logVal=(float*)malloc(sizeof(float)*logSize);

    for(int i=0;i<logSize;i++){
        if(i==0){
            logVal[0]=-9999;
        }else{
            logVal[i]=log2((float)i);
        }
        // printf("%.2f ",logVal[i]);
    }

    // ready the value needed
    hipMalloc(&distributedArray,arraySize);
    hipMalloc(&distributedEntropyArray,entropySize);
    hipMalloc(&distributedLogVal,logSize*sizeof(float));

    
    hipMemcpy(distributedArray,array,arraySize,hipMemcpyHostToDevice);
    hipMemcpy(distributedEntropyArray,entropy,entropySize,hipMemcpyHostToDevice);
    hipMemcpy(distributedLogVal,logVal,logSize*sizeof(float),hipMemcpyHostToDevice);

    // launch the kernel
    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    dim3 dimGrid((width+dimBlock.x-1)/dimBlock.x,(height+dimBlock.y-1)/dimBlock.y);

    entropy_kernel<<<dimGrid,dimBlock>>>(distributedArray,distributedEntropyArray,distributedLogVal,
        width,height,window_size);
    
    hipMemcpy(entropy,distributedEntropyArray,entropySize,hipMemcpyDeviceToHost);


    // free memory
    hipFree(distributedArray);
    hipFree(distributedEntropyArray);
    hipFree(distributedLogVal);

    free(logVal);

    return;
}

int main(int argc,char* argv[]){
    if(argc<4){
        fprintf(stderr,"Please input the size of the 2-dimension array.\n");
        exit(1);
    }

    int width=atoi(argv[1]),height=atoi(argv[2]),window_size=atoi(argv[3]);
    // printf("%d %d %d\n",width,height,window_size);

    int *array=(int*)malloc(sizeof(int)*width*height);
    float* entropy=(float*)malloc(sizeof(float)*width*height);

    // initial value between [0-15]
    srand(time(NULL));
    for(int i=0;i<width*height;i++){
        array[i]=(int)(rand()/(float)RAND_MAX*15);
        entropy[i]=0;
    }

    // // define the file 
    // int fd;
    // char INPUTFILE[]="random_input.txt";
    // char OUTPUTFILE[]="entropy.txt";

    // // store input
    // fd=open(INPUTFILE,O_CREAT|O_WRONLY,0644);
    // if(fd<0){
    //     fprintf(stderr,"open input file error");
    //     exit(1);
    // }

    // for(int i=0;i<width*height;i++){
    //     char buffer[10];
    //     sprintf(buffer,"%d\t",array[i]);
    //     write(fd,buffer,strlen(buffer));

    //     if((i+1) % height==0){
    //         write(fd,"\n",1);
    //     }
    // }

    // close(fd);

    struct timeval start,end;
    //begining timer
    gettimeofday(&start,NULL);

    // calculate
    allocate(array,entropy,width,height,window_size);

    // end timer;
    gettimeofday(&end,NULL);
    double time_using=1e6*(end.tv_sec-start.tv_sec)+end.tv_usec-start.tv_usec;
    printf("time using:%lfs\n",time_using/1e6);

    // // store output
    // fd=open(OUTPUTFILE,O_CREAT|O_WRONLY,0644);
    // if(fd<0){
    //     fprintf(stderr,"open input file error");
    //     exit(1);
    // }

    // for(int i=0;i<width*height;i++){
    //     char buffer[10];
    //     sprintf(buffer,"%.2f\t",entropy[i]);
    //     write(fd,buffer,strlen(buffer));

    //     if((i+1) % height==0){
    //         write(fd,"\n",1);
    //     }
    // }

    // close(fd);

    // free memory
    free(array);
    free(entropy);

    return 0;
}