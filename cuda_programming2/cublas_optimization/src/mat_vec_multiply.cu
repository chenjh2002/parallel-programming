#include "hip/hip_runtime.h"
#include "mat_vec_multiply.hh"
#include <hipblas.h>

// #define BLOCK_SIZE 32

__global__ void matrixVectorMutiply1Dim(float *A, float *b, float *c,int row,int col){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < row){
        float sum = 0.0;
        for(int j =0; j < col; j++){
            sum += A[i * col + j] * b[j];
        }
        c[i] = sum;
    }
}

// __global__ void matrixVectorMutiply2Dim(float** A, float *b, float* c, int row, int col){
//     int x = blockIdx.x * blockDim.x + threadIdx.x;

// }

void matrixVectorMutiplyExecute1Dim(float* mat, float* vec, float* res, int row, int col,int blockSize){
    // value pass to GPU memory
    float *distributedMat = NULL;
    float *distributedVec = NULL;
    float *distributedRes = NULL;

    // calculate szie
    int matSize = row * col *sizeof(float);
    int vecSize = col * sizeof(float);
    int resSize = row * sizeof(float);

    // prepare the value
    hipMalloc(&distributedMat, matSize);
    hipMalloc(&distributedVec, vecSize);
    hipMalloc(&distributedRes, resSize);

    hipMemcpy(distributedMat, mat, matSize, hipMemcpyHostToDevice);
    hipMemcpy(distributedVec, vec, vecSize, hipMemcpyHostToDevice);
    hipMemcpy(distributedRes, res, resSize, hipMemcpyHostToDevice);

    // // launch the kernel
    // dim3 dimBlock(blockSize);
    // dim3 dimGrid((row + blockSize - 1) / blockSize);

    // matrixVectorMutiply1Dim<<<dimBlock,dimGrid>>>(distributedMat, distributedVec, distributedRes, row, col);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0f, beta = 0.0f;

    hipblasSgemv(handle, HIPBLAS_OP_N, row, col, &alpha, distributedMat, row, distributedVec, col, &beta, distributedRes, 1);

    // remove data from device to host
    hipMemcpy(res,distributedRes,resSize,hipMemcpyDeviceToHost);

    // free memory
    hipFree(distributedMat);
    hipFree(distributedVec);
    hipFree(distributedRes);

    return;
}

void produceRandomArray(float* mat,int size){
    for(int i=0; i < size; i++){
        mat[i]=rand() / (float)RAND_MAX * 125; 
    }
}

float* globalAllocaMat1Dim(int row, int col, bool isRand){
    float* mat = (float*)malloc(row * col * sizeof(float));
    int matSize = row * col;

    if(isRand){
        produceRandomArray(mat, matSize);
    }else{
        memset(mat, 0, matSize * sizeof(float));
    }

    return mat;
}

float** globalAllocaMat2Dim(int row, int col, bool isRand){
    float* mat = (float*)malloc(row * col * sizeof(float));
    int matSize = row * col;

    if(isRand){
        produceRandomArray(mat, matSize);
    }else{
        memset(mat, 0, matSize * sizeof(float));
    }

    float** mat2Dim = (float**)malloc(row * sizeof(float*));
    for(int i=0; i < row; i++){
        mat2Dim[i] = &mat[i * (int)col];
    }

    return mat2Dim;
}

float* globalAllocaVec(int length, bool isRand){
    float* vec = (float*)malloc(length * sizeof(float));
    if(isRand){
        produceRandomArray(vec, length);
    }else{
        memset(vec, 0, length * sizeof(float));
    }

    return vec;
}


void globalFree(float* mat,float *vec, float *result){
    free(mat);
    free(vec);
    free(result);

    return;
}

// int main(int argc, char * argv[]){
//     int row =256, col = 256;
//     float * mat, * vec, *res;

//     mat = globalAllocaMat1Dim(row, col, true);
//     vec = globalAllocaVec(col, true);
//     res = globalAllocaVec(row, false);

//     matrixVectorMutiplyExecute1Dim(mat, vec, res, row, col,32);

//     for(int i = 0; i < row; i++){
//         printf("%.5f\t",res[i]);
//     }
//     printf("\n");

//     globalFree(mat, vec, res);

//     return 0;
// }
