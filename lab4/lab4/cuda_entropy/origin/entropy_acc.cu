#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <openacc.h>

#define ROWS 1000
#define COLS 1000

void calculate_entropy(double *data, double *entropy, int rows, int cols) {
    #pragma acc data copyin(data[0:rows*cols]) copyout(entropy[0:cols])
    {
        #pragma accparallel loop collapse(2)
        for (int i = 0; i < cols; i++) {
            for (int j = 0; j < rows; j++) {
                int index = j * cols + i;
                double value = data[index];
                entropy[i] -= value * log2(value);
            }
        }
    }
}

int main() {
    double *data = (double*) malloc(sizeof(double) * ROWS * COLS);
    double *entropy = (double*) malloc(sizeof(double) * COLS);

    // Initialize data
    for (int i = 0; i < ROWS * COLS; i++) {
        data[i] = (double) rand() / RAND_MAX;
    }

    // Calculate entropy
    calculate_entropy(data, entropy, ROWS, COLS);

    // Print results
    for (int i = 0; i < COLS; i++) {
        printf("Column %d entropy: %f\n", i, entropy[i]);
    }

    free(data);
    free(entropy);

    return 0;
